#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <iostream>
#include <cmath>
#include "cutlass/cutlass.h"
#include "cutlass/gemm/device/gemm.h"
#include "cutlass/gemm/device/gemm_array.h"
#include "cutlass/gemm/device/gemm_batched.h"
#include "cutlass/layout/matrix.h"
#include "kernels_calls.h"
#include "wrapper.h"
#define CEIL(M, N) (((M) + (N)-1) / (N)) 

using namespace std;

// cublas matrix multiplication kernel T precision
template <typename T>
T* call_cublasGemm(const T* h_A, const T* h_B, int N)
{
    // handle creation for cublas gemm
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Allocate memory on GPU
    T* d_A;
    T* d_B;
    T* d_C;
    T* h_C = new T[N * N];

    hipMalloc(&d_A, N * N * sizeof(T));
    hipMalloc(&d_B, N * N * sizeof(T));
    hipMalloc((void**)&d_C, N * N * sizeof(T));

    // Copy data from host to device
    hipMemcpy(d_A, h_A, N * N * sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * N * sizeof(T), hipMemcpyHostToDevice);

    const T alpha = 1.0;
    const T beta = 0.0;
    cublasGemm_w(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha, d_B, N, d_A,
                 N, &beta, d_C, N);
    hipMemcpy(h_C, d_C, N * N * sizeof(T), hipMemcpyDeviceToHost);

    hipblasDestroy(handle);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return h_C;
}
template double* call_cublasGemm(const double* h_A, const double* h_B, int N);
template float* call_cublasGemm(const float* h_A, const float* h_B, int N);

double* call_DgemmGlobalMemory(const double* h_A, const double* h_B, int N)
{
    double* h_C = new double[N * N];

    // Allocate memory on GPU
    double* d_A;
    double* d_B;
    double* d_C;

    hipMalloc(&d_A, N * N * sizeof(double));
    hipMalloc(&d_B, N * N * sizeof(double));
    hipMalloc(&d_C, N * N * sizeof(double));
    // Copy data from host to device
    hipMemcpy(d_A, h_A, N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * N * sizeof(double), hipMemcpyHostToDevice);

    // Calculate the grid and block dimensions for CUDA kernels
    dim3 block(32 * 32);
    dim3 grid(CEIL(N,32), CEIL(N,32));

    // Call the global memory double precision kernel
    globalMemoryDgemm<<<grid, block>>>(d_A, d_B, d_C, N);

    // Copy results back from device to host
    hipMemcpy(h_C, d_C, N * N * sizeof(double), hipMemcpyDeviceToHost);

    // Free memory on GPU
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return h_C;
}

float* call_SgemmGlobalMemory(const float* h_A, const float* h_B, int N)
{
    float* h_C = new float[N * N];

    // Allocate memory on GPU
    float* d_A;
    float* d_B;
    float* d_C;

    hipMalloc(&d_A, N * N * sizeof(float));
    hipMalloc(&d_B, N * N * sizeof(float));
    hipMalloc(&d_C, N * N * sizeof(float));
    // Copy data from host to device
    hipMemcpy(d_A, h_A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * N * sizeof(float), hipMemcpyHostToDevice);

    // Calculate the grid and block dimensions for CUDA kernels
    dim3 block(32 * 32);
    dim3 grid(CEIL(N,32), CEIL(N,32));

    // Call the global memory float precision kernel
    globalMemorySgemm<<<grid, block>>>(d_A, d_B, d_C, N);

    // Copy results back from device to host
    hipMemcpy(h_C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);

    // Free memory on GPU
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return h_C;
}

double* call_DgemmSharedMemory(const double* h_A, const double* h_B, int N)
{
    double* h_C = new double[N * N];

    // Allocate memory on GPU
    double* d_A;
    double* d_B;
    double* d_C;

    hipMalloc(&d_A, N * N * sizeof(double));
    hipMalloc(&d_B, N * N * sizeof(double));
    hipMalloc(&d_C, N * N * sizeof(double));
    // Copy data from host to device
    hipMemcpy(d_A, h_A, N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * N * sizeof(double), hipMemcpyHostToDevice);

    // Calculate the grid and block dimensions for CUDA kernels
    dim3 block(32 * 32);
    dim3 grid(CEIL(N,32), CEIL(N,32));

    // Call the global memory double precision kernel
    SharedMemoryDgemm<<<grid, block>>>(d_A, d_B, d_C, N);

    // Copy results back from device to host
    hipMemcpy(h_C, d_C, N * N * sizeof(double), hipMemcpyDeviceToHost);

    // Free memory on GPU
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return h_C;
}

float* call_SgemmSharedMemory(const float* h_A, const float* h_B, int N)
{
    float* h_C = new float[N * N];

    // Allocate memory on GPU
    float* d_A;
    float* d_B;
    float* d_C;

    hipMalloc(&d_A, N * N * sizeof(float));
    hipMalloc(&d_B, N * N * sizeof(float));
    hipMalloc(&d_C, N * N * sizeof(float));
    // Copy data from host to device
    hipMemcpy(d_A, h_A, N * N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * N * sizeof(float), hipMemcpyHostToDevice);

    // Calculate the grid and block dimensions for CUDA kernels

    dim3 block(256);
    dim3 grid(CEIL(N,128), CEIL(N,128));


    // Call the global memory float precision kernel
    SharedMemorySgemm<<<grid, block>>>(d_A, d_B, d_C, N);

    // Copy results back from device to host
    hipMemcpy(h_C, d_C, N * N * sizeof(float), hipMemcpyDeviceToHost);

    // Free memory on GPU
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return h_C;
}


// cublas Batched matrix multiplication kernel T precision
template <typename T>
T* call_cublasGemmBatched(const T* h_A, const T* h_B, int N, int BATCH_SIZE)
{
    T* h_C = new T[N * N * BATCH_SIZE];
    T *d_A, *d_B, *d_C;
    size_t size = sizeof(T) * N * N * BATCH_SIZE;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    hipMemcpy(d_C, h_C, size, hipMemcpyHostToDevice);
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    T *A_array[BATCH_SIZE], *B_array[BATCH_SIZE];
    T* C_array[BATCH_SIZE];
    {
        for (int j = 0; j < BATCH_SIZE; ++j) {
            A_array[j] = d_A + j * N;
            B_array[j] = d_B + j * N;
            C_array[j] = d_C + j * N;
        }
    }
    const T **d_A_array, **d_B_array;
    T** d_C_array;
    hipMalloc((void**)&d_A_array, BATCH_SIZE * sizeof(T*));
    hipMalloc((void**)&d_B_array, BATCH_SIZE * sizeof(T*));
    hipMalloc((void**)&d_C_array, BATCH_SIZE * sizeof(T*));
    hipMemcpy(d_A_array, A_array, BATCH_SIZE * sizeof(T*),
               hipMemcpyHostToDevice);
    hipMemcpy(d_B_array, B_array, BATCH_SIZE * sizeof(T*),
               hipMemcpyHostToDevice);
    hipMemcpy(d_C_array, C_array, BATCH_SIZE * sizeof(T*),
               hipMemcpyHostToDevice);

    const T alpha = 1.0;
    const T beta = 0.0;

    int lda = N * BATCH_SIZE;
    int ldb = N * BATCH_SIZE;
    int ldc = N * BATCH_SIZE;

    cublasBatchedGemm_w(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha,
                        d_B_array, ldb, d_A_array, lda, &beta, d_C_array, ldc,
                        BATCH_SIZE);
    hipblasDestroy(handle);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_A_array);
    hipFree(d_B_array);
    hipFree(d_C_array);

    return h_C;
}
template double* call_cublasGemmBatched(const double* h_A, const double* h_B,
                                        int N, int BATCH_SIZE);
template float* call_cublasGemmBatched(const float* h_A, const float* h_B,
                                       int N, int BATCH_SIZE);

//
template <typename T>
T* call_BatchedGemmGlobalMemory(const T* h_A, const T* h_B, int N,
                                int BATCH_SIZE)
{
    T *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, N * N * BATCH_SIZE * sizeof(T));
    hipMalloc((void**)&d_B, N * N * BATCH_SIZE * sizeof(T));
    hipMalloc((void**)&d_C, N * N * BATCH_SIZE * sizeof(T));

    // Transfer input matrices from host to GPU
    hipMemcpy(d_A, h_A, N * N * BATCH_SIZE * sizeof(T),
               hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * N * BATCH_SIZE * sizeof(T),
               hipMemcpyHostToDevice);

    dim3 block_dim(8, 128);
    dim3 grid_dim((N + block_dim.x - 1) / block_dim.x,
                  (N * BATCH_SIZE + block_dim.y - 1) / block_dim.y);

    T* h_C = new T[N * N * BATCH_SIZE];

    GlobalMemoryBatchedGemm<<<grid_dim, block_dim>>>(d_A, d_B, d_C, N,
                                                     BATCH_SIZE);

    hipDeviceSynchronize();
    // Transfer result matrix from GPU to host
    hipMemcpy(h_C, d_C, N * N * BATCH_SIZE * sizeof(T),
               hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return h_C;
}
template double* call_BatchedGemmGlobalMemory(const double* h_A,
                                              const double* h_B, int N,
                                              int BATCH_SIZE);
template float* call_BatchedGemmGlobalMemory(const float* h_A, const float* h_B,
                                             int N, int BATCH_SIZE);

template <typename T>
T* call_BatchedGemmSharedMemory(const T* h_A, const T* h_B, int N,
                                int BATCH_SIZE)
{
    T *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, N * N * BATCH_SIZE * sizeof(T));
    hipMalloc((void**)&d_B, N * N * BATCH_SIZE * sizeof(T));
    hipMalloc((void**)&d_C, N * N * BATCH_SIZE * sizeof(T));

    // Transfer input matrices from host to GPU
    hipMemcpy(d_A, h_A, N * N * BATCH_SIZE * sizeof(T),
               hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, N * N * BATCH_SIZE * sizeof(T),
               hipMemcpyHostToDevice);


    dim3 block_dim(32, 32);
    dim3 grid_dim((N + block_dim.x - 1) / block_dim.x,
                  (N * BATCH_SIZE + block_dim.y - 1) / block_dim.y);
    T* h_C = new T[N * N * BATCH_SIZE];

    SharedMemoryBatchedGemm<<<grid_dim, block_dim>>>(d_A, d_B, d_C, N,
                                                     BATCH_SIZE);

    hipDeviceSynchronize();
    // Transfer result matrix from GPU to host
    hipMemcpy(h_C, d_C, N * N * BATCH_SIZE * sizeof(T),
               hipMemcpyDeviceToHost);

    // Free GPU memory
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return h_C;
}
template double* call_BatchedGemmSharedMemory(const double* h_A,
                                              const double* h_B, int N,
                                              int BATCH_SIZE);
template float* call_BatchedGemmSharedMemory(const float* h_A, const float* h_B,
                                             int N, int BATCH_SIZE);


template <typename T>
T* call_CutlassBatchedGemm(const T* h_A, const T* h_B, int N, int BATCH_SIZE)
{
    T* A;
    T* B;
    T* C;
    T alpha = 1.0;
    T beta = 2.0;

    hipMalloc(&A, N * N * BATCH_SIZE * sizeof(T));
    hipMalloc(&B, N * N * BATCH_SIZE * sizeof(T));
    hipMalloc(&C, N * N * BATCH_SIZE * sizeof(T));

    hipMemcpy(A, h_A, N * N * BATCH_SIZE * sizeof(T), hipMemcpyHostToDevice);
    hipMemcpy(B, h_B, N * N * BATCH_SIZE * sizeof(T), hipMemcpyHostToDevice);

    using Gemm =
        cutlass::gemm::device::GemmBatched<T, cutlass::layout::ColumnMajor, T,
                                           cutlass::layout::ColumnMajor, T,
                                           cutlass::layout::ColumnMajor>;

    Gemm gemm_op;

    cutlass::Status status = gemm_op({{N, N, N},
                                      {B, N * BATCH_SIZE},
                                      N,
                                      {A, N * BATCH_SIZE},
                                      N,
                                      {C, N * BATCH_SIZE},
                                      N,
                                      {C, N * BATCH_SIZE},
                                      N,
                                      {alpha, beta},
                                      BATCH_SIZE});

    T* h_C = new T[N * N * BATCH_SIZE];

    hipMemcpy(h_C, C, N * N * BATCH_SIZE * sizeof(T), hipMemcpyDeviceToHost);

    hipFree(A);
    hipFree(B);
    hipFree(C);


    return h_C;
}
template double* call_CutlassBatchedGemm(const double* h_A, const double* h_B,
                                         int N, int BATCH_SIZE);
template float* call_CutlassBatchedGemm(const float* h_A, const float* h_B,
                                        int N, int BATCH_SIZE);