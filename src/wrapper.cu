#include "wrapper.h"

// Template specialization for float
template <>
void cublasGemm_w<float>(hipblasHandle_t handle, hipblasOperation_t transa,
                         hipblasOperation_t transb, int m, int n, int k,
                         const float* alpha, const float* A, int lda,
                         const float* B, int ldb, const float* beta, float* C,
                         int ldc)
{
    hipblasSgemm(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C,
                ldc);
}


// Template specialization for double
template <>
void cublasGemm_w<double>(hipblasHandle_t handle, hipblasOperation_t transa,
                          hipblasOperation_t transb, int m, int n, int k,
                          const double* alpha, const double* A, int lda,
                          const double* B, int ldb, const double* beta,
                          double* C, int ldc)
{
    hipblasDgemm(handle, transa, transb, m, n, k, alpha, A, lda, B, ldb, beta, C,
                ldc);
}
// Template specialization for float
template <>
void cublasBatchedGemm_w<float>(hipblasHandle_t handle, hipblasOperation_t transa,
                                hipblasOperation_t transb, int m, int n, int k,
                                const float* alpha, const float* Aarray[],
                                int lda, const float* Barray[], int ldb,
                                const float* beta, float* Carray[], int ldc,
                                int batchCount)
{
    hipblasSgemmBatched(handle, transa, transb, m, n, k, alpha, Aarray, lda,
                       Barray, ldb, beta, Carray, ldc, batchCount);
}


template <>
void cublasBatchedGemm_w<double>(hipblasHandle_t handle,
                                 hipblasOperation_t transa,
                                 hipblasOperation_t transb, int m, int n, int k,
                                 const double* alpha, const double* Aarray[],
                                 int lda, const double* Barray[], int ldb,
                                 const double* beta, double* Carray[], int ldc,
                                 int batchCount)
{
    hipblasDgemmBatched(handle, transa, transb, m, n, k, alpha, Aarray, lda,
                       Barray, ldb, beta, Carray, ldc, batchCount);
}
